#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include ""

#define err(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename FUNC>
__global__ void f_kernel(float* a, float* b, int size, FUNC func) 
{ 
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x)
      b[i] = func(a[i]);
}

template <typename FUNC>
std::vector<float> f(std::vector<float> const& a, FUNC func)
{
    std::vector<float> b(a.size());
    const int bsize = a.size() * 4;
    float* da;
    float* db;
    err(hipMalloc(&da, bsize));
    err(hipMalloc(&db, bsize));
    err(hipMemcpy(da, a.data(), bsize, hipMemcpyHostToDevice));
    f_kernel<<<256, 256>>>(da, db, a.size(), func);
    err(hipDeviceSynchronize());

    err(hipMemcpy(b.data(), db, bsize, hipMemcpyDeviceToHost));
    err(hipFree(da));
    err(hipFree(db));
    return b;
}

// I have to explicit instantiate but don't know how to do this for lambdas
